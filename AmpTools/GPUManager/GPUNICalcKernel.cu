#include "hip/hip_runtime.h"
//******************************************************************************
// This file is part of AmpTools, a package for performing Amplitude Analysis
//
// Copyright Trustees of Indiana University 2010, all rights reserved
//
// This software written by Matthew Shepherd, Ryan Mitchell, and
//                  Hrayr Matevosyan at Indiana University, Bloomington
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
// 1. Redistributions of source code must retain the above copyright
//    notice and author attribution, this list of conditions and the
//    following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright
//    notice and author attribution, this list of conditions and the
//    following disclaimer in the documentation and/or other materials
//    provided with the distribution.
// 3. Neither the name of the University nor the names of its contributors
//    may be used to endorse or promote products derived from this software
//    without specific prior written permission.
//
// Creation of derivative forms of this software for commercial
// utilization may be subject to restriction; written permission may be
// obtained from the Trustees of Indiana University.
//
// INDIANA UNIVERSITY AND THE AUTHORS MAKE NO REPRESENTATIONS OR WARRANTIES,
// EXPRESS OR IMPLIED.  By way of example, but not limitation, INDIANA
// UNIVERSITY MAKES NO REPRESENTATIONS OR WARRANTIES OF MERCANTABILITY OR
// FITNESS FOR ANY PARTICULAR PURPOSE OR THAT THE USE OF THIS SOFTWARE OR
// DOCUMENTATION WILL NOT INFRINGE ANY PATENTS, COPYRIGHTS, TRADEMARKS,
// OR OTHER RIGHTS.  Neither Indiana University nor the authors shall be
// held liable for any liability with respect to any claim by the user or
// any other party arising from use of the program.
//******************************************************************************

#include "GPUCustomTypes.h"
#include "stdio.h"

__global__ void
ni_calc_kernel( int nElements, GDouble* pfDevNICalc,
                GDouble* pfDevAmps, GDouble* pfDevWeights,
                int nEvents, int nTrueEvents )
{

  // used shared memory block for amplitude indices and results
  extern __shared__ int s[];

  // get addresses of arrays for the two indices in shared memory
  // and also the result
  GDouble* result = (GDouble*)s;
  int* iIndex = (int*)&result[2*nElements];
  int* jIndex = &(iIndex[nElements]);

  // this is the integer array of indices on the device
  int* piIndexDev = (int*)&pfDevNICalc[2*nElements];

  // this is the thread index in the block -- use to try
  // to parallelize block-level operations:
  int iThread = threadIdx.x + GPU_BLOCK_SIZE_X * threadIdx.y;

  // have each thread copy a portion of the index array from
  // device memory to shared memory and zero out the place for
  // the result in shared memory
  // (for nElements large, this paralellizes the setup)
  int i = iThread;
  while( i < 2*nElements ){

     iIndex[i] = piIndexDev[i];
     result[i] = 0;
     i += GPU_BLOCK_SIZE_SQ;
  } 
  
  __syncthreads();
  
  // this is the overall event index
  int iEvt = iThread +
            ( blockIdx.x + blockIdx.y * gridDim.x ) * GPU_BLOCK_SIZE_SQ;

  if( iEvt < nTrueEvents ) // do not compute for the padding events
  for( int i = 0; i < nElements; ++i ){
  
    // these are the indices to the relevant amplitudes in the amplitude array
    int aInd = 2*iEvt + 2*nEvents*iIndex[i];
    int bInd = 2*iEvt + 2*nEvents*jIndex[i];

    GDouble thisRe, thisIm = 0;

    thisRe = pfDevWeights[iEvt] * (
                     pfDevAmps[aInd]   * pfDevAmps[bInd]  +
                     pfDevAmps[aInd+1] * pfDevAmps[bInd+1] );

    atomicAdd_block( &result[2*i], thisRe );

    if( aInd == bInd ) continue; // diagonal elements are real

    thisIm = pfDevWeights[iEvt] * (
                       pfDevAmps[aInd+1] * pfDevAmps[bInd] -
                       pfDevAmps[aInd]   * pfDevAmps[bInd+1] );

    atomicAdd_block( &result[2*i+1], thisIm );
  }
  
  __syncthreads();
  
  // now accumulate global device memory with the result...
  // again this is done in parallel with each thread adding
  // the real or imaginary part of a term in the array of
  // nElements

  i = iThread;
  while( i < 2*nElements ){

     atomicAdd( &pfDevNICalc[i], result[i] );
     i += GPU_BLOCK_SIZE_SQ;
  }   
}


extern "C" void GPU_ExecNICalcKernel( dim3 dimGrid, dim3 dimBlock,
       	   			      unsigned int sharedSize,
                                      int nElements, GDouble* pfDevNICalc,
                                      GDouble* pfDevAmps, GDouble* pfDevWeights,
                                      int nEvents, int nTrueEvents )
{
  ni_calc_kernel<<< dimGrid, dimBlock, sharedSize >>>
     ( nElements, pfDevNICalc, pfDevAmps, pfDevWeights, nEvents, nTrueEvents );
}
